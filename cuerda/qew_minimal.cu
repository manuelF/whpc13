#include "hip/hip_runtime.h"
/*
=============================================================================================
A. B. Kolton

Segundo Encuentro Nacional de Computación de Alto Rendimiento para Aplicaciones Científicas
7 al 10 de Mayo de 2013.
=============================================================================================

Este programita simula la dinámica de una cuerdita elastica en un medio desordenado en la GPU, 
usando la librería Thrust, y la libreria Random123. Es una versión reducida de la que 
usamos con E. Ferrero y S. Bustingorry para la publicación:

Phys. Rev. E 87, 032122 (2013)
Nonsteady relaxation and critical exponents at the depinning transition
http://pre.aps.org/abstract/PRE/v87/i3/e032122
http://arxiv.org/abs/1211.7275

Y la explicación resumida del código esta en el material suplementario de la revista:
http://pre.aps.org/supplemental/PRE/v87/i3/e032122


Sin resolver los TODO ya se puede compilar con "make", y larga algunos timmings.
Genera dos ejecutables a la vez, uno de CUDA (corre en GPU) y otro de openMP (corre en CPU multicore).

OBJETIVOS:
- Practicar el manejo básico de la biblioteca Thrust.
- Practicar el manejo básico de la biblioteca Random123.
- Comparar performances CPU vs GPU.
- Aprender a combinar herramientas para resolver una ecuación diferencial 
parcial estocástica con desorden congelado.

EJERCICIOS:	
- Levantar los TODO.
- Para los mas expertos: Como mejoraría la performance del codigo?
*/


#include "timer.h"
#include <cmath>
#include <fstream>
#include <iostream>

/* counter-based random numbers */
// http://www.thesalmons.org/john/random123/releases/1.06/docs/
#include <Random123/philox.h> // philox headers
#include <Random123/u01.h>    // to get uniform deviates [0,1]
typedef r123::Philox2x32 RNG; // particular counter-based RNG


/* algunos headers de la libreria thrust */
// https://github.com/thrust/thrust/wiki/Quick-Start-Guide
// https://github.com/thrust/thrust/wiki/Documentation
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>

/* parámetros del problema */
#ifndef TAMANIO
#define L	4096   // numero de partículas/monomeros
#else
#define L	TAMANIO   // numero de partículas/monomeros
#endif
#ifndef TIEMPORUN
#define TRUN	100000       // numero de iteraciones temporales 
#else
#define TRUN	TIEMPORUN    // numero de iteraciones temporales 
#endif
#ifndef TPROP
#define TPROP	1000 // intervalo entre mediciones
#endif

#define F0	0.12       // fuerza uniforme sobre la interface/polímero 
#define Dt	0.1       // paso de tiempo
#define TEMP	0.003     // temperatura	
#define D0	1.0	  // intensidad del desorden
#define SEED 	12345678 // global seed RNG (quenched noise)
#define SEED2 	12312313 // global seed#2 RNG (thermal noise)

// para evitar poner "thrust::" a cada rato all llamar sus funciones
using namespace thrust;

// precisón elegida para los números reales
typedef double REAL;

// para generar números aleatorios gausianos a partir de dos uniformes
// http://en.wikipedia.org/wiki/Box%E2%80%93Muller_transform
__device__
REAL box_muller(RNG::ctr_type r_philox)
{
	// transforma el philox number a dos uniformes en (0,1]
 	REAL u1 = u01_open_closed_32_53(r_philox[0]);
  	REAL u2 = u01_open_closed_32_53(r_philox[1]);

  	REAL r = sqrt( -2.0*log(u1) );
  	REAL theta = 2.0*M_PI*u2;
	return r*sin(theta);    			
}

#ifdef PRINTCONFS
// para imprimir configuraciones cada tanto...
std::ofstream configs_out("configuraciones.dat");
void print_configuration(device_vector<REAL> &u, device_vector<REAL> &Ftot, float velocity, int size)
{
	long i=0;
	int every = int(L*1.0/size);
	for(i=1;i<u.size()-1;i+=every)
	{
		configs_out << u[i] << " " << Ftot[i] << " " << velocity << std::endl;
	}
	configs_out << "\n\n";
}
#endif

// functor usado para calcular la rugosidad 
struct roughtor: public thrust::unary_function<REAL,REAL>
{
    REAL u0; // un "estado interno" del functor	
    roughtor(REAL _u0):u0(_u0){};	
    __device__
    REAL operator()(REAL u)
    {	
	return (u-u0)*(u-u0);
    }
};	



/////////////////////////////////////////////////////////////////////////////
// FUNCTORS usados en los algoritmos TRANSFORM

struct fuerza
{
    RNG rng;       // random number generator
    long tiempo;   // parámetro
    REAL noiseamp; // parámetro
		
    fuerza(long _t):tiempo(_t)
    {
	noiseamp=sqrt(TEMP/Dt);
    }; 

    __device__
    REAL operator()(tuple<long,REAL,REAL,REAL> tt)
    {	
	// thread/particle id
	uint32_t tid=get<0>(tt); 

	// keys and counters 
    	RNG::ctr_type c={{}};
    	RNG::key_type k={{}};
	RNG::ctr_type r;

	k[0]=tid; 	  //  KEY = {threadID} 
	c[1]=SEED; 	  // COUNTER[1] = {a fijar mas tarde, GLOBAL SEED}

	// LAPLACIAN
	REAL um1=get<1>(tt);
	REAL u=get<2>(tt);
	REAL up1=get<3>(tt);
	REAL laplaciano = um1 + up1 - 2.*u;

	// DISORDER
#ifndef NODISORDER
	REAL firstRN, secondRN;
	int U=int(u);
	c[0]=uint32_t(U); // COUNTER={U,GLOBAL SEED}
	r = rng(c, k);
	firstRN=(u01_closed_closed_32_53(r[0])-0.5); // alternativa: box_muller(r);
	c[0]=uint32_t(U+1); // COUNTER={U+1,GLOBAL SEED}
	r = rng(c, k);
	secondRN=(u01_closed_closed_32_53(r[0])-0.5); // alternativa: box_muller(r);
	REAL quenched_noise = D0*(firstRN - (firstRN-secondRN)*(u-U));// linearly interpolated force
#endif
	// THERMAL NOISE
	REAL thermal_noise;

// TODO: agregar -DFINITETEMPERATURE en el Makefile para agregar 
// fluctuaciones termicas pero antes corregir el FIXME siguiente! 
#ifdef FINITETEMPERATURE
	// FIXME: Lo de abajo tiene un error grave!!. Corrijalo.
	c[0] = tid; // COUNTER = {tid, GLOBAL SEED #2} 
	c[1] = SEED2; // para evitar correlaciones entre el ruido térmico y el desorden congelado...
	r = rng(c, k);
	thermal_noise = noiseamp*box_muller(r);    			
#else
	thermal_noise=0.0;
#endif
	// Fuerza total en el monómero tid
	return (laplaciano+quenched_noise+thermal_noise+F0);
  }
};


// Explicit forward Euler step: lo mas simple que hay 
// (pero ojo con el paso de tiempo que no sea muy grande!)
struct euler
{
    __device__
    REAL operator()(REAL u_old, REAL force)
    {	
	return (u_old + force*Dt);
    }
};	


#ifdef OMP
#include <omp.h>
#endif

/////////////////////////////////////////////////////////////////////////////
int main(){

	#ifdef OMP
	std::cout << "#conociendo el host, OMP threads = " << omp_get_max_threads() << std::endl;
	#endif		

	/* containers e iteradores */

	// posiciones de los monómeros: 
	// Notar que alocamos dos elementos de mas para usarlos como "halo"
	// esto nos permite fijar las condiciones de borde facilmente, por ejemplo periódicas: u[0]=u[L]; u[L+1]=u[1];
	device_vector<REAL> u(L+2);

	// dos iteradores para definir el rango de interes para aplicar algoritmos 
	// (el +-1 nos permite descartar el halo en los algoritmos) 
	device_vector<REAL>::iterator u_it0 = u.begin()+1;
	device_vector<REAL>::iterator u_it1 = u.end()-1;

	// Si necesita el puntero "crudo" al array para pasarselo a un kernel de CUDA C/C++:
	REAL * u_raw_ptr = raw_pointer_cast(&u[1]);

	// container de fuerza total: 
	device_vector<REAL> Ftot(L); 
	// el rango de interés definido por los iteradores 
	device_vector<REAL>::iterator Ftot_it0 = Ftot.begin();
	device_vector<REAL>::iterator Ftot_it1 = Ftot.end();

	// alguna condición inicial chata (arbitraria)
	fill(u_it0,u_it1,0.0); 


	// simple (GPU/CPU) timer (curiosear el common/timer.h)
	timer t;
	double timer_fuerzas_elapsed=0.0;
	double timer_euler_elapsed=0.0;
	double timer_props_elapsed=0.0;

	// file para guardar algunas propiedades dependientes del tiempo
	#ifndef OMP
	std::ofstream propsout("someprops.dat");
	#else
	std::ofstream propsout("someprops_omp.dat");
	#endif

	// loop temporal
	//functor_fuerza fuerza(0);

	device_vector<REAL> u_old(L+2);
	for(long n=0;n<TRUN;n++)
	{
		// Impone PBC en el "halo"		
		u[0]=u[L];u[L+1]=u[1];


		t.restart(); // para cronometrar el tiempo de la siguiente transformación
		
		// Fuerza en cada monómero calculada concurrentemente en la GPU: 
		// Ftot(X)= laplacian + disorder + thermal_noise + F0, X=0,..,L-1
		// mirar el functor "fuerza" mas arriba... 
		// Notar: los iteradores de interés estan agrupado con un "fancy" zip_iterator, 
		// ya que transform no soporta mas de dos secuencias como input.
		// Notar: make_counting_iterator es otro "fancy" iterator, 
		// que simula una secuencia que en realidad no existe en memoria (implicit sequences).
		// https://github.com/thrust/thrust/wiki/Quick-Start-Guide
		// http://thrust.github.io/doc/group__fancyiterator.html
		//fuerza.set_time(n);
		transform(
			make_zip_iterator(make_tuple(
			make_counting_iterator<long>(0),u_it0-1,u_it0,u_it0+1
			)),
			make_zip_iterator(make_tuple(
			make_counting_iterator<long>(L),u_it1-1,u_it1,u_it1+1
			)),
			Ftot_it0,
			fuerza(n)
		);

		timer_fuerzas_elapsed+=t.elapsed();

		// Explicit forward Euler step, implementado en paralelo en la GPU: 
		// u(X,n) += Ftot(X,n) Dt, X=0,...,L-1
		// Mirar el functor "euler" mas arriba...
		// Notar: no hace falta zip_iterator, ya que transform si soporta hasta dos secuencias de input
        if(n%TPROP==0)
        {
            u_old=u;
        }

		t.restart();
		transform(
			u_it0,u_it1,Ftot_it0,u_it0, 
			euler()
		);
		timer_euler_elapsed+=t.elapsed();

		// algunas propiedades de interés, calculadas cada TPROP
		if(n%TPROP==0){		
			t.restart();
			
			/* TODO:
			   usando algoritmos REDUCE 
			   [ http://thrust.github.io/doc/group__reductions.html#gacf5a4b246454d2aa0d91cda1bb93d0c2 ]
			   calcule la velocidad media de la interface
			   y la posición del centro de masa de la interface 
			   HINT:
			   REAL velocity = reduce(....)/L; //center of mass velocity
			   REAL center_of_mass = reduce(....)/L; // center of mass position
			*/

			   REAL velocity = (reduce(           u.begin()+1,               u.end()-1,               0.0

               )
               +
               reduce(
               u_old.begin()+1,
               u_old.end()-1,
               0.0
               )
               )
               /L; //center of mass velocity
               
               /* Velocidad = delta Posicion / delta tiempo
                  SUM ( U(n)-U(n-1) ) /  Dt
                

               */
               

			   REAL center_of_mass = reduce(....)/L; // center of mass position
	               /*Centro de masas = Promedio de fuerzas
                  


               */
               

		/* TODO: 
			   usando el algoritmo TRANSFORM_REDUCE, 
			   [ http://thrust.github.io/doc/group__transformed__reductions.html#ga087a5af8cb83647590c75ee5e990ef66 ]
			   el functor "roughtor" arriba definido, 
			   y la posición del centro de masa "ucm" calculada en el TODO anterior, 
			   calcule la rugosidad (mean squared width) de la interface:
			   roughness := Sum_X [u(X)-ucm]^2 /L
			   HINT:
			   REAL roughness = transform_reduce(...,...,roughtor(center_of_mass),0.0,thrust::plus<REAL>());
			*/
			timer_props_elapsed+=t.elapsed();
	

			/* TODO: descomentar para que imprima la velocidad media, centro de masa, y rugosidad 
			   calculadas en los otros "TODOes", en el file "someprops.dat" */	
			//propsout << velocity << " " << center_of_mass << " " << roughness << std::endl;
			//propsout.flush();
			
			/* TODO:
			 Descomente -DPRINTCONFS en el Makefile, y recompile, para que imprima la posición 
			 y velocidad de 128 (o lo que quiera) particulas de la interface de tamanio L (una de cada L/128) 
			 en pantalla (descomente con cuidado, que imprime mucho!. Solo para hacer un "intuitive debugging")
			*/
			#ifdef PRINTCONFS
			print_configuration(u,Ftot,velocity,128);
			#endif
		}

		/* TODO: visualization!.
		   Si hay algun experimentado en el uso de, por ejemplo openGL, que se le ocurra como 
		   visualizar la línea en tiempo real en una máquina con el monitor conectado a la placa,
		   le agradeceré me enseñe :-). */
	}

	// resultados del timming
	double total_time = (timer_fuerzas_elapsed+timer_euler_elapsed+timer_props_elapsed); 

	std::cout << "L= " << L << " TRUN= " << TRUN << std::endl; 

	std::cout << "Forces calculation -> " << 1e3 * timer_fuerzas_elapsed << " miliseconds (" 
		  << int(timer_fuerzas_elapsed*100/total_time) << "%)" << std::endl;

	std::cout << "Euler step -> " << 1e3 * timer_euler_elapsed << " miliseconds (" 
		  << int(timer_euler_elapsed*100/total_time) << "%)" << std::endl;

	std::cout << "Properties -> " << 1e3 * timer_props_elapsed << " miliseconds (" 
		  << int(timer_props_elapsed*100/total_time) << "%)" << std::endl;

	std::cout << "Total -> " << 1e3 * total_time << " miliseconds (100%)" << std::endl;

	return 0;
}
