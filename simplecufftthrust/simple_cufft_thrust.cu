#include "hip/hip_runtime.h"
/*
=============================================================================================
A. B. Kolton

Segundo Encuentro Nacional de Computación de Alto Rendimiento para Aplicaciones Científicas
7 al 10 de Mayo de 2013.
=============================================================================================

Este programita transforma Fourier una señal unidimensional, almacenada en GPU, usando cufft.
Se puede compilar y correr directamente, midiendo el tiempo de GPU, e imprimiendo la transformada.

$ nvcc simple_cufft_thrust.cu -lcufft -arch=sm_20 -o simple_cufft

OBJETIVOS:
- Practicar el manejo básico de la librería cuFFT.
- Practicar el manejo básico de la librería Thrust.
- Practicar la interoperabilidad cuFFT-Thrust.

EJERCICIOS:
- Utilizar simple y doble precisión, y diferentes tamaños. Comparar performances.
- Levantar los TODO.
- Importante: ¿Cómo están ordenadas las frecuencias de la transformada? 
*/


/* algunos headers de la libreria thrust */
// https://github.com/thrust/thrust/wiki/Documentation
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>

#include <iostream>
#include <fstream>
#include "cutil.h"	// CUDA_SAFE_CALL, CUT_CHECK_ERROR
#include "timer.h"


// CUFFT include http://docs.nvidia.com/cuda/cufft/index.html
#include <hipfft/hipfft.h>

using namespace std;

/* + Array Size N: use only powers of 2 */
#ifndef TAMANIO
#define N 1048576
#else
#define N TAMANIO
#endif

#define DOUBLE_PRECISION

#ifdef DOUBLE_PRECISION
typedef hipfftDoubleReal REAL;
typedef hipfftDoubleComplex COMPLEX;
#else
typedef hipfftReal REAL;
typedef hipfftComplex COMPLEX;
#endif

/* Parametros de la senial */
#define A1 4
#define A2 6
#define T1 N/4
#define T2 N/8
struct FillSignal
{
	__device__ __host__ 
	REAL operator()(unsigned tid)
    	{	
		// ponga aqui su funcion preferida...
		return A1*2.0*cosf(2*M_PI*tid*T1/(float)N) + A2*2.0*sinf(2*M_PI*tid*T2/(float)N);
    	}
};

#ifdef OMP
#include <omp.h>
#endif

///////////////////////////////////////////////////////////////////////////
int main(void) {

	#ifdef OMP
	std::cout << "#conociendo el host, OMP threads = " << omp_get_max_threads() << std::endl;
	#endif

	// Un container de thrust para guardar el input real en GPU 
	thrust::device_vector<REAL> D_input(N);

	// toma el raw_pointer del array de input, para pasarselo a CUFFT luego
	REAL *d_input = thrust::raw_pointer_cast(&D_input[0]);

	// Un container de thrust para guardar el ouput complejo en GPU = transformada del input 
	int Ncomp=N/2+1;
	thrust::device_vector<COMPLEX> D_output(Ncomp);

	// toma el raw_pointer del array de output, para pasarselo a CUFFT luego
	COMPLEX *d_output = thrust::raw_pointer_cast(&D_output[0]); 

	// crea el plan de transformada de cuFFT
	#ifdef DOUBLE_PRECISION
	hipfftHandle plan_d2z;
	CUFFT_SAFE_CALL(hipfftPlan1d(&plan_d2z,N,HIPFFT_D2Z,1));
	#else
	hipfftHandle plan_r2c;
	CUFFT_SAFE_CALL(hipfftPlan1d(&plan_r2c,N,HIPFFT_R2C,1));
	#endif

	//lleno array de tamanio N con la senial, a travez del functor "FillSignal"
	thrust::transform(thrust::make_counting_iterator(0),thrust::make_counting_iterator(N),D_input.begin(),FillSignal());

	/* ---- Start ---- */
	// un timer para GPU
	timer t;
	t.restart();

	//Transforma Fourier ejecutando el plan
	#ifdef DOUBLE_PRECISION
	CUFFT_SAFE_CALL(hipfftExecD2Z(plan_d2z, d_input, d_output));
	#else
	CUFFT_SAFE_CALL(hipfftExecR2C(plan_r2c, d_input, d_output));
	#endif

	double t_elapsed=t.elapsed();
	/* ---- Stop ---- */

	// declara un vector para copiar/guardar la transformada en el host:
	thrust::host_vector<COMPLEX> H_output=D_output;

	/* Imprime la transformada */
	cout << "# Tamanio del array = " << N << endl;
	cout << "# Tiempo de GPU " << 1e3*t_elapsed << " miliseconds" << endl;
#ifdef IMPRIMIR
	ofstream transformada_out("transformada.dat");
	for(int j = 0 ; j < Ncomp ; j++){
		transformada_out << COMPLEX(H_output[j]).x << " " << COMPLEX(H_output[j]).y << endl;
	}
    transformada_out.close();
#endif

// TODO: Verifique que el resultado sea correcto, por ejemeplo, usando seniales cuya transformada conoce analiticamente 
// o verificando simetrias, etc. Preste atencion al ordenamiento de las frecuencias...
// HINT: sin(x)=[e^{ix}-e^{-ix}]/2i , cos(x)=[e^{ix}+e^{-ix}]/2 y mire la formula de la antitransformada (clase)

// TODO: 
// Agregue planes para realizar la antitransformada de la senial con CUFFT (contemple los casos double y float)
//	#ifdef DOUBLE_PRECISION
//	#else
//	#endif
	#ifdef DOUBLE_PRECISION
	hipfftHandle plan_z2d;
	CUFFT_SAFE_CALL(hipfftPlan1d(&plan_z2d,N,HIPFFT_Z2D,1));
	#else
	hipfftHandle plan_c2r;
	CUFFT_SAFE_CALL(hipfftPlan1d(&plan_c2r,N,HIPFFT_C2R,1));
	#endif

// TODO: 
// Declare/aloque un container de Thrust para guardar la antitransformada, y el raw_pointer para pasar a CUFFT
	thrust::device_vector<REAL> D_AntiTransformed(N);

	// toma el raw_pointer del array de output, para pasarselo a CUFFT luego
	REAL *d_Anti = thrust::raw_pointer_cast(&D_AntiTransformed[0]); 


// TODO:
// Ejecute los planes cuFFT de la antitransformada (contemple los casos double y float)
//	#ifdef DOUBLE_PRECISION
//	#else
//	#endif

	#ifdef DOUBLE_PRECISION
	CUFFT_SAFE_CALL(hipfftExecZ2D(plan_z2d, d_output, d_Anti));
	#else
	CUFFT_SAFE_CALL(hipfftExecC2R(plan_c2r, d_output, d_Anti));
	#endif


// TODO: 
// Declare/aloque dos containers de Thrust: uno para guardar la antitransformada, y otro para el input original, en el host
// y copie los respectivos contenidos del device al host

	thrust::host_vector<REAL> Original_input=D_input;
	thrust::host_vector<REAL> AntiTransformed_output=D_AntiTransformed;


// TODO:
// Imprima en un file el input original y la antitransformada de la transformada, para comparar
#ifdef IMPRIMIR
	ofstream comparativa_out("comparativa.dat");
	for(int j = 0 ; j < N ; j++){
		comparativa_out << Original_input[j] << "\t " << AntiTransformed_output[j] << endl;
	}
    comparativa_out.close();
#endif
	#ifdef DOUBLE_PRECISION
    hipfftDestroy(plan_d2z);
    hipfftDestroy(plan_z2d);

    #else
    hipfftDestroy(plan_r2c);
    hipfftDestroy(plan_c2r);
    #endif
	return 0;
}
